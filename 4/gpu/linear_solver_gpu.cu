#include "hip/hip_runtime.h"
#include "linear_solver_gpu.h"

__global__ void create_b(const double *, const double *, double *);
__global__ void create_a(const double *, double *, int);
__global__ void resolve_row(const double *, const double *, double *, double *, double *, double, int);

hipError_t resolve_system_gpu(const double* left, const double* right, double* result, int dimension, double epsilon)
{
    double* dev_left = 0;
    double* dev_right = 0;
    double* dev_res = 0;
    double* dev_b_column = 0;
    double* dev_a_matrix = 0;
    double* dev_prev_step = 0;
    double* dev_tmp = 0;

    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);


    if (cudaStatus != hipSuccess)
    {
        return cudaStatus;
    }

    // Allocate GPU buffers
    cudaStatus = hipMalloc((void**)&dev_left, dimension * dimension * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        return cudaStatus;
    }

    cudaStatus = hipMalloc((void**)&dev_right, dimension * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        hipFree(dev_left);
        return cudaStatus;
    }

    cudaStatus = hipMalloc((void**)&dev_res, dimension * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        hipFree(dev_left);
        hipFree(dev_right);
        return cudaStatus;
    }


    cudaStatus = hipMalloc((void**)&dev_b_column, dimension * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        hipFree(dev_left);
        hipFree(dev_right);
        hipFree(dev_res);
        return cudaStatus;
    }

    cudaStatus = hipMalloc((void**)&dev_a_matrix, dimension * dimension * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        hipFree(dev_left);
        hipFree(dev_right);
        hipFree(dev_res);
        hipFree(dev_b_column);
        return cudaStatus;
    }


    cudaStatus = hipMalloc((void**)&dev_prev_step, dimension * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        hipFree(dev_left);
        hipFree(dev_right);
        hipFree(dev_res);
        hipFree(dev_b_column);
        hipFree(dev_a_matrix);
        return cudaStatus;
    }

    cudaStatus = hipMalloc((void**)&dev_tmp, dimension * sizeof(double));
    if (cudaStatus != hipSuccess) {
        hipFree(dev_left);
        hipFree(dev_right);
        hipFree(dev_res);
        hipFree(dev_b_column);
        hipFree(dev_a_matrix);
        hipFree(dev_prev_step);
        return cudaStatus;
    }

    // Copy input matrixes from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_left, left, dimension * dimension * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        hipFree(dev_left);
        hipFree(dev_right);
        hipFree(dev_res);
        hipFree(dev_b_column);
        hipFree(dev_a_matrix);
        hipFree(dev_prev_step);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_right, right, dimension * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        hipFree(dev_left);
        hipFree(dev_right);
        hipFree(dev_res);
        hipFree(dev_b_column);
        hipFree(dev_a_matrix);
        hipFree(dev_prev_step);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_prev_step, result, dimension * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        hipFree(dev_left);
        hipFree(dev_right);
        hipFree(dev_res);
        hipFree(dev_b_column);
        hipFree(dev_a_matrix);
        hipFree(dev_prev_step);
        return cudaStatus;
    }

    // Start with creation of b_column
    create_b <<<1, dimension>>> (dev_left, dev_right, dev_b_column);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        hipFree(dev_left);
        hipFree(dev_right);
        hipFree(dev_res);
        hipFree(dev_b_column);
        hipFree(dev_a_matrix);
        hipFree(dev_prev_step);
        return cudaStatus;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        hipFree(dev_left);
        hipFree(dev_right);
        hipFree(dev_res);
        hipFree(dev_b_column);
        hipFree(dev_a_matrix);
        hipFree(dev_prev_step);
        return cudaStatus;
    }


    dim3 threadsPerBlock(dimension, dimension);
    dim3 blocksPerGrid(1, 1);
    if (dimension * dimension > 512)
    {
        threadsPerBlock.x = 512;
        threadsPerBlock.y = 512;
        blocksPerGrid.x = (unsigned int)(double(dimension) / double(threadsPerBlock.x));
        blocksPerGrid.y = (unsigned int)(double(dimension) / double(threadsPerBlock.y));
    }

    create_a <<<blocksPerGrid, threadsPerBlock >>> (dev_left, dev_a_matrix, dimension);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        hipFree(dev_left);
        hipFree(dev_right);
        hipFree(dev_res);
        hipFree(dev_b_column);
        hipFree(dev_a_matrix);
        hipFree(dev_prev_step);
        return cudaStatus;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        hipFree(dev_left);
        hipFree(dev_right);
        hipFree(dev_res);
        hipFree(dev_b_column);
        hipFree(dev_a_matrix);
        hipFree(dev_prev_step);
        return cudaStatus;
    }


    resolve_row <<<blocksPerGrid, threadsPerBlock >>> (dev_a_matrix, dev_b_column,
                                                       dev_prev_step, dev_tmp, dev_res, epsilon,
                                                       dimension);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        hipFree(dev_left);
        hipFree(dev_right);
        hipFree(dev_res);
        hipFree(dev_b_column);
        hipFree(dev_a_matrix);
        hipFree(dev_prev_step);
        return cudaStatus;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        hipFree(dev_left);
        hipFree(dev_right);
        hipFree(dev_res);
        hipFree(dev_b_column);
        hipFree(dev_a_matrix);
        hipFree(dev_prev_step);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(result, dev_res, dimension * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        hipFree(dev_left);
        hipFree(dev_right);
        hipFree(dev_res);
        hipFree(dev_b_column);
        hipFree(dev_a_matrix);
        hipFree(dev_prev_step);
        return cudaStatus;
    }

    hipFree(dev_left);
    hipFree(dev_right);
    hipFree(dev_res);
    hipFree(dev_b_column);
    hipFree(dev_a_matrix);
    hipFree(dev_prev_step);
    return cudaStatus;
}

__global__ void create_b(const double* left, const double* right, double* b_column)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int dimension = blockDim.x; // размерность определяется размером блока

    if (idx < dimension)
    {
        b_column[idx] = right[idx] / left[idx * dimension + idx];
    }
}


__global__ void create_a(const double* left, double* a_matrix, int dimension)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row == col)
    {
        a_matrix[row * dimension + col] = 0;
    }
    else
    {
        a_matrix[row * dimension + col] = -left[row * dimension + col] / left[row * dimension + row];
    }
}


__global__ void resolve_row(const double* a_matrix, const double *b_column, double* prev_step, double* tmp, double* result, double epsilon, int N)
{
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;

    double tmpSum;

    if (ROW < N) {
        // each thread computes one element of the block sub-matrix
        while(1)
        {
            tmpSum = 0;
            for (int i = 0; i < N; i++) {
                tmpSum += a_matrix[ROW * N + i] * prev_step[i];
            }
            result[ROW] = b_column[ROW] + tmpSum;

            if (result[ROW] > prev_step[ROW])
            {
                if (result[ROW] - prev_step[ROW] < epsilon)
                {
                    break;
                }
            }
            else
            {
                if (prev_step[ROW] - result[ROW] < epsilon)
                {
                    break;
                }
            }

            prev_step[ROW] = result[ROW];
        }
    }
}